#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2018, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include "tutorial.h"


rtBuffer<BasicLight> lights;

rtDeclareVariable(float3, geometric_normal, attribute geometric_normal, );
rtDeclareVariable(float3, shading_normal, attribute shading_normal, );
rtDeclareVariable(float4, obj_color, attribute obj_color, );
rtDeclareVariable(float4, ambient_light_color, , );
rtDeclareVariable(float, phong_exp, , );

rtDeclareVariable(PerRayData_radiance, prd_radiance, rtPayload, );
rtDeclareVariable(PerRayData_shadow, prd_shadow, rtPayload, );

rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );
rtDeclareVariable(float, t_hit, rtIntersectionDistance, );
rtDeclareVariable(uint2, launch_index, rtLaunchIndex, );

rtDeclareVariable(float, scene_epsilon, , );
rtDeclareVariable(rtObject, top_object, , );
rtDeclareVariable(rtObject, top_shadower, , );


//
// Pinhole camera implementation
//
rtDeclareVariable(float3, eye, , );
rtDeclareVariable(float3, U, , );
rtDeclareVariable(float3, V, , );
rtDeclareVariable(float3, W, , );
rtDeclareVariable(float3, bad_color, , );
rtBuffer<float4, 2>              output_buffer;

RT_PROGRAM void pinhole_camera()
{
	size_t2 screen = output_buffer.size();

	float2 d = make_float2(launch_index) / make_float2(screen) * 2.f - 1.f;
	float3 ray_origin = eye;
	float3 ray_direction = normalize(d.x * U + d.y * V + W);

	optix::Ray ray(ray_origin, ray_direction, RADIANCE_RAY_TYPE, scene_epsilon);

	PerRayData_radiance prd;
	prd.importance = 1.f;
	prd.depth = 0;

	rtTrace(top_object, ray, prd);

	output_buffer[launch_index] = prd.result;
}

//
// Returns solid color for miss rays
//
rtDeclareVariable(float3, bg_color, , );
RT_PROGRAM void miss()
{
	
	prd_radiance.result = make_float4(bg_color, 0.01f);
}

RT_PROGRAM void closest_hit_radiance()
{
	// intersection vectors
	const float3 h = ray.origin + t_hit * ray.direction;            // hitpoint
	const float3 n = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, shading_normal)); // normal
	const float3 i = ray.direction;                                            // incident direction

	float reflection = 1.0f;
	float4 result = make_float4(0.0f);
	float3 refraction_color = make_float3(1);

	float3 beer_attenuation;
	if (dot(n, ray.direction) > 0) {
		// Beer's law attenuation
		float3 extinction_constant = make_float3(log(.80f), log(.89f), log(.75f));
		beer_attenuation = exp(extinction_constant * t_hit);
	}
	else {
		beer_attenuation = make_float3(1);
	}
	float max_depth = 100;
	// refraction
	if (prd_radiance.depth < max_depth)
	{
		float3 t;                                                            // transmission direction
		float refraction_index = 1.4f;
		if (refract(t, i, n, refraction_index))
		{

			// check for external or internal reflection
			float cos_theta = dot(i, n);
			if (cos_theta < 0.0f)
				cos_theta = -cos_theta;
			else
				cos_theta = dot(t, n);

			reflection = fresnel_schlick(cos_theta, 3, .1, 1.0f);

			float importance = prd_radiance.importance * (1.0f - reflection) * optix::luminance(refraction_color * beer_attenuation);
			if (importance > .01f) {
				optix::Ray ray(h, t, RADIANCE_RAY_TYPE, scene_epsilon);
				PerRayData_radiance refr_prd;
				refr_prd.depth = prd_radiance.depth + 1;
				refr_prd.importance = importance;

				rtTrace(top_object, ray, refr_prd);
				result += make_float4((1.0f - reflection) * refraction_color, 1) * refr_prd.result;
			}
			else {
				result += make_float4((1.0f - reflection) * refraction_color, 1) * obj_color;
			}
		}
		// else TIR
	}
	float3 reflection_color = make_float3(1);
	// reflection
	if (prd_radiance.depth < max_depth)
	{
		float3 r = reflect(i, n);

		float importance = prd_radiance.importance * reflection * optix::luminance(reflection_color * beer_attenuation);
		if (importance > .01f) {
			optix::Ray ray(h, r, RADIANCE_RAY_TYPE, scene_epsilon);
			PerRayData_radiance refl_prd;
			refl_prd.depth = prd_radiance.depth + 1;
			refl_prd.importance = importance;

			rtTrace(top_object, ray, refl_prd);
			result += make_float4(reflection * reflection_color, 1) * refl_prd.result;
		}
		else {
			result += make_float4(reflection * reflection_color, 1) * obj_color;
		}
	}

	result = result * make_float4(beer_attenuation, 1);
	if(result.w > .5)
		rtPrintf("%f\n", result.w);
	prd_radiance.result = result;
}


//
// Returns shading normal as the surface shading result
// 
RT_PROGRAM void closest_hit_radiance1()
{
	float3 world_geo_normal = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, geometric_normal));
	float3 world_shade_normal = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, shading_normal));
	float3 ffnormal = faceforward(world_shade_normal, -ray.direction, world_geo_normal);
	float3 hit_point = ray.origin + t_hit * ray.direction;
	float4 color = make_float4(0);

	float3 beer_attenuation;
	if (dot(world_shade_normal, ray.direction) > 0) {
		// Beer's law attenuation
		float3 extinction_constant = make_float3(log(.80f), log(.89f), log(.75f));
		
		beer_attenuation  = exp(extinction_constant * t_hit);
	}
	else {
		beer_attenuation = make_float3(1);
	}

	for (int i = 0; i < lights.size(); i++) {
		BasicLight light = lights[i];
		float3 L = normalize(light.pos - hit_point);
		float nDl = dot(ffnormal, L);

		if (nDl > 0) {
			PerRayData_shadow shadow_prd;
			shadow_prd.attenuation = make_float3(1.0f);
			float Ldist = length(light.pos - hit_point);
			optix::Ray shadow_ray(hit_point, L, SHADOW_RAY_TYPE,
				scene_epsilon, Ldist);
			rtTrace(top_shadower, shadow_ray, shadow_prd);
			float3 light_attenuation = shadow_prd.attenuation;
			if (fmaxf(light_attenuation) > 0.0f) {
				float3 Lc = light.color * light_attenuation;
				//color += obj_color * make_float4(nDl * light.color, 1.0f);

				float3 H = normalize(L - ray.direction);
				float nDh = dot(ffnormal, H);
				if (false)
					color += obj_color * make_float4(Lc * pow(nDh, phong_exp), 1.0);
			}
		}
	}
	float max_depth = 10;
	float3 refraction_color = make_float3(1);
	float importance = prd_radiance.importance * optix::luminance(refraction_color * beer_attenuation);
	//rtPrintf("%f\n", prd_radiance.importance);
	if (prd_radiance.depth < max_depth && importance > .01f) {
		PerRayData_radiance transmit_prd;
		transmit_prd.depth = prd_radiance.depth + 1;
		transmit_prd.importance = importance;
		Ray trans_ray(hit_point, ray.direction, RADIANCE_RAY_TYPE, scene_epsilon);

		rtTrace(top_object, trans_ray, transmit_prd);
		float transmittence_var = 1/(transmit_prd.depth + 50);
		color += transmittence_var * transmit_prd.result;
	}
	//rtPrintf("%f, %f, %f\n", color.w, beer_attenuation.y, beer_attenuation.z);
	prd_radiance.result = make_float4(beer_attenuation, 1.0) * color;
}

RT_PROGRAM void any_hit_shadow() {
	float3 world_normal = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, shading_normal));
	float nDi = fabs(dot(world_normal, ray.direction));

	prd_shadow.attenuation *= 1 - fresnel_schlick(nDi, 5);
	rtIgnoreIntersection();
	//rtTerminateRay();
}


//
// Set pixel to solid color upon failur
//
RT_PROGRAM void exception()
{
	output_buffer[launch_index] = make_float4(bad_color, 1.0f);
}
