#include "hip/hip_runtime.h"
#include <optix.h>
#include <optixu/optixu_math_namespace.h>
#include <optix_device.h>
#include "volume_render.h"



using namespace optix;

struct PerRayData {
	;
};

rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );
rtDeclareVariable(float, t_hit, rtIntersectionDistance, );
rtDeclareVariable(uint2, launch_index, rtLaunchIndex, );

rtBuffer<float4, 2> amplitude_buffer;


rtDeclareVariable(int, volumeTextureId1, , );
rtDeclareVariable(int, rayTextureId, , );
rtDeclareVariable(int, normalTextureId1, , );
rtDeclareVariable(int, max_volume, , );
rtDeclareVariable(int, depth_mask_id, , );

rtDeclareVariable(float3, texcoord, attribute texcoord, );
rtDeclareVariable(float3, front_hit_point, attribute front_hit_point, );
rtDeclareVariable(float3, back_hit_point, attribute back_hit_point, );

// information about the volume
rtDeclareVariable(float3, box_min, , );	// opposite corners of the volume
rtDeclareVariable(float3, box_max, , );
rtDeclareVariable(float3, v1, , );		// edges of the plane in which a slice is put, has been scaled by 1/dot(v1, v1)
rtDeclareVariable(float3, v2, , );
rtDeclareVariable(float3, v3, , );

// step size
rtDeclareVariable(float, volumeRaytraceStepSize, , );

// lighting_stuff

rtDeclareVariable(float, ambientStrength, , );
rtDeclareVariable(float2, lightDirP, , );
rtDeclareVariable(float3, lightDir, , );
rtDeclareVariable(float2, sincosLightTheta, , );
rtDeclareVariable(float3, CameraDir, , );
rtDeclareVariable(float3, HalfwayVec, , );
rtDeclareVariable(float3, sincosCameraDirTheta, , );
rtDeclareVariable(float2, CameraDirP, , );
rtDeclareVariable(float2, HalfwayVecP, , );

rtDeclareVariable(float2, sincosHalfwayTheta, , );
rtDeclareVariable(float, specularStrength, , );
rtDeclareVariable(float, shininess, , );
rtDeclareVariable(float, diffuseStrength, , );


rtDeclareVariable(float, zFar, , );
rtDeclareVariable(float, zNear, , );

rtDeclareVariable(float2, IsoValRange, , );
rtDeclareVariable(float3, ShadingTerms, , );
rtDeclareVariable(float4, BubbleTerms, , );
rtDeclareVariable(float, tune, , );
rtDeclareVariable(float3, color1, , );
rtDeclareVariable(float3, color2, , );
rtDeclareVariable(float3, color3, , );
rtDeclareVariable(float3, color4, , );
rtDeclareVariable(float3, color5, , );
rtDeclareVariable(float3, color6, , );
rtDeclareVariable(int, numTex, , );
rtDeclareVariable(int, enabledColors, , );
rtDeclareVariable(float4, intersectionColor, , );
rtDeclareVariable(float, debug, , );
rtDeclareVariable(float, increment, , );
rtDeclareVariable(float, step_mod, , );


RT_PROGRAM void dummy() {
	//rtPrintf("%d, %d\n", launch_index.x, launch_index.y);
	amplitude_buffer[launch_index] = make_float4(.7, 0, .9, .6);
}



rtDeclareVariable(float3, hg_normal, , );	// normalized
//For use on two vectors with radius 1 in spherical coordinates
inline float sdot(float2 a, float2 b) {
	return  sin(a.x) * sin(b.x) * cos(a.y - b.y) + cos(a.x) * cos(b.x);
}
//For use on a sincos vector and the normal vector. Uses precomputed sines and cosines
inline float sdot(float2 sincosa, float2 sincosnorm, float a, float phi) {
	return sincosa.x * sincosnorm.x * cos(a - phi) + sincosa.y * sincosnorm.y;
}

RT_PROGRAM void closest_hit() {
	//amplitude_buffer[launch_index] = make_float4(1, 0, 1, .5);
	//return;
	float2 sample;
	/*
	float max_theta = -3.15f;
	float min_theta = 3.15f;
	for (int i = 0; i < 512; i++) {
		for (int j = 0; j < 512; j++) {
			for (int k = 0; k < 512; k++) {
				sample = optix::rtTex3D<float2>(normalTextureId1, i/512.0f, j/512.0f, k/512.0f);
				//float theta = sample.x * M_PIf;
				float theta = sample.y * M_PIf * 2 - M_PIf;
				if (theta < min_theta) {
					min_theta = theta;
				}
				else if (theta > max_theta) {
					max_theta = theta;
				}
			}
		}
	}
	rtPrintf("%f, %f\n", min_theta, max_theta);
	*/
	const float3 fhp = front_hit_point;
	
	const float3 bhp = back_hit_point;
	float total_distance = (sqrtf(dot(bhp - fhp, bhp - fhp)));
	float3 color_composited = make_float3(0.f, 0.f, 0.f);
	float opaque_composited = 0.f;
	//float epsilon = .1f * volumeRaytraceStepSize;
	//bool show_spec = true;
	float depth = optix::rtTex2D<float>(depth_mask_id, launch_index.x / (float)amplitude_buffer.size().x, (amplitude_buffer.size().y - launch_index.y) / (float)amplitude_buffer.size().y) * 2.0f - 1;
	float distance = 0;
	
	//if ((zFar + zNear - depth * (zFar - zNear)) > 0) {
	distance = 2.0 * zNear * zFar / (zFar + zNear - depth * (zFar - zNear));
	
	float distance_so_far = 0;
	//rtPrintf("%f\n", total_distance);
	while (total_distance > distance_so_far) {
		//rtPrintf("%f\n", distance_so_far);
		float3 texPoint = fhp + (distance_so_far) * ray.direction;// +(epsilon) * (optix::rtTex3D<float>(random_texture, launch_index.x / amplitude_buffer.size().x, launch_index.y / amplitude_buffer.size().y, s / num_steps));
		
		float vol_u = dot(texPoint - box_min, v1);
		float vol_v = dot(texPoint - box_min, v2);
		float vol_w = dot(texPoint - box_min, v3);
		//rtPrintf("%f, %f, %f\n", v1.x, v2.y, v3.z);
		float3 show = texPoint - box_min;

		float volume_scalar;
		if (length(texPoint - ray.origin) > distance) {
			//color_composited = make_float3(1, 0, 0);
			//color_composited = make_float3(0, 0, 0);
			//opaque_composited = 0;
			//flag = true;
			//rtPrintf("%f\n", length(texPoint-box_min));
			//amplitude_buffer[launch_index] = make_float4(length(texPoint-ray.origin)/50.0f, length(texPoint-ray.origin) /50.0f, length(texPoint-ray.origin) /50.0f, 1.0f);
			amplitude_buffer[launch_index] = make_float4((color_composited.x), (color_composited.y), (color_composited.z), (opaque_composited));
			return;
		}
		
		
		
		float4 color;
		bool flag = false;
		bool lighting_enabled = enabledColors &(1<<5);
		float volume_max = optix::rtTex2D<float>(max_volume, vol_u, vol_v) - increment * vol_w;
		distance_so_far += volumeRaytraceStepSize * (1 + step_mod * max(abs(volume_max - IsoValRange.x), abs(volume_max - IsoValRange.y)));
		float4 voxel_val_tf;

		if (volume_max <= IsoValRange.y) {
			//rtPrintf("%f\n", volume_scalar);
			continue;
		}
		for (int i = 0; i < numTex; i++) {
			if (!(enabledColors & (1 << i)))
				continue;

			bool shade_intersection = false;
			switch (i) {
			case 0:
				color = make_float4(color1, 1.0f);
				//color = make_float4(0, 0, 1, 1.0f);
				break;
			case 1:
				color = make_float4(color2, 1.0f);
				break;
			case 2:
				color = make_float4(color3, 1.0f);
				break;
			case 3:
				color = make_float4(color4, 1.0f);
				break;
			case 4:
				color = make_float4(color5, 1.0f);
				break;
			default:
				color = make_float4(1.0);
			}
			volume_scalar = optix::rtTex3D<float>(volumeTextureId1, vol_u, vol_v, i / (float)numTex) - increment * vol_w;
			float4 voxel_val_tf;
			if (volume_scalar <= IsoValRange.y && volume_scalar >= IsoValRange.x) {
				voxel_val_tf = color;
				if (flag == true) {
					voxel_val_tf = intersectionColor;
					shade_intersection = true;
					//return;
				}
				flag = true;
				//voxel_val_tf = make_float4(fabs(normal.x), fabs(normal.y), fabs(normal.z), .99);
				//rtPrintf("%f, %f, %f\n", voxel_val_tf.x, voxel_val_tf.y, voxel_val_tf.z);
			}
			else {
				//voxel_val_tf = make_float4(0, 0, 0, 0);
				continue;
			}

			sample = optix::rtTex3D<float2>(normalTextureId1, vol_u, vol_v, i / 5.0f);
			
			//float top_val = .565;
			//float top_val = .215;
			//float bottom_val = .555;
			//float bottom_val = .2;
			// = optix::rtTex2D<float4>(transferFunction_texId, volume_scalar, volume_scalar);

			
			float phi = sample.x * M_PIf;
			float theta = sample.y * M_PIf * 2 - M_PIf;

			


			float3 color_self = make_float3(0);
			float opaque_self = 0;

			float sinphi = sin(phi);
			float3 normal = make_float3(sinphi * cos(theta), sinphi * sin(theta), cos(phi));
			//normal.z = 1 - increment;
			//normal = normalize(normal);
			float2 normalP = make_float2(phi, theta);
			float2 sincosnorm = make_float2(sin(theta), cos(theta));

			//sin(theta1)sin(theta2)cos(phi1 - pih2) + cos(theta1)cos(theta2)
			//float diffuse = diffuseStrength * fmax(0, sin(theta) * sincosLightTheta.x * cos(phi - lightDirP.x) + cos(theta) * sincosLightTheta.y);
			//float diffuse = diffuseStrength * fmax(0, sdot(lightDirP, normalP));
			float diffuse = diffuseStrength * fmax(0, sdot(sincosLightTheta, sincosnorm, lightDirP.y, phi));
			//float3 viewDir = CameraDir;




			//float spec = specularStrength * pow(fmax(sin(theta) * sincosHalfwayTheta.x * cos(phi - sincosHalfwayTheta.x) + cos(theta) * sincosHalfwayTheta.y, 0), shininess);
			//float spec = specularStrength * pow(fabs(sdot(normalP, HalfwayVecP)), shininess);
			float spec = specularStrength * pow(fabs(sdot(sincosHalfwayTheta, sincosnorm, HalfwayVecP.y, phi)), shininess);
			//rtPrintf("%f, %f\n", HalfwayVecP.x, HalfwayVecP.y);
			if (lighting_enabled) {
				color_self = ambientStrength * make_float3(voxel_val_tf) + diffuse * make_float3(voxel_val_tf) + spec * make_float3(1, 1, 1);
			}
			else {
				color_self = make_float3(fabs(normal.x), fabs(normal.y), fabs(normal.z));;
			}
			

			float bubble_coefficient = 1 - (fabs(dot(ray.direction, normal)));
			//float bubble_coefficient = 1 - (fabs(sdot(CameraDirP, normalP)));

			if (!shade_intersection) {
				if (voxel_val_tf.w > 1e-5) {
					if (bubble_coefficient > BubbleTerms.x) {
						bubble_coefficient = BubbleTerms.x;
					}
					else if (bubble_coefficient < BubbleTerms.y) {
						bubble_coefficient = BubbleTerms.y;
					}

					float norm = (((bubble_coefficient - BubbleTerms.y) / (BubbleTerms.x - BubbleTerms.y)));
					bubble_coefficient = (norm * (BubbleTerms.z - BubbleTerms.w)) + BubbleTerms.w;
					voxel_val_tf.w = ShadingTerms.x;

					//voxel_val_tf.w = 1.0f;
				}
				//bubble_coefficient /= 4;
				//bubble_coefficient += .75;
				//bubble_coefficient = 0 ;
				//if(spec != 0)
					//rtPrintf("%f\n", spec);

				opaque_self = voxel_val_tf.w + (abs(ShadingTerms.y) * bubble_coefficient + ((ShadingTerms.z) * spec));// 0.5f);
				if (ShadingTerms.y > 0) {
					color_self *= powf(1 - bubble_coefficient, 1 / tune);
				}
			}
			else {
				opaque_self = intersectionColor.w;
			}

			/*
			if (opaque_self > .01) {
				opaque_self = 1.0f;
				//rtPrintf("%f, %f, %f\n", normal.x, normal.y, normal.z);
				if (fabs(normal.x) < 0.01 && fabs(normal.y) < 0.01 && fabs(normal.z) < 0.01) {
					opaque_self = 0;
				}
			}
			*/
			//opaque_self = 1.f - powf(1.f - opaque_self, opacity_correction);
			// amp = amp_in + (1-opaque) * amp_self
			// opqaue = opaque_in + (1-opqaue) * opqaue_self
			//rtPrintf("%f, %f, %f\n", color_self.x, color_self.y, color_self.z);
			color_composited += ( (1.f - opaque_composited) * color_self * opaque_self);
			opaque_composited +=  (1.f - opaque_composited) * opaque_self;
			//color_composited = color_self;


			//if (counter	 < 300 && opaque_composited > 1e-3 ) {
				//opacities[counter] = opaque_self;
				//counter++;
			//}
			
			if (opaque_composited > 0.9) {
				amplitude_buffer[launch_index] = make_float4((color_composited.x), (color_composited.y), (color_composited.z), (opaque_composited));
				return;
			}
		}
	}
	
	amplitude_buffer[launch_index] = make_float4((color_composited.x), (color_composited.y), (color_composited.z), (opaque_composited));
}


rtDeclareVariable(uint2, element_hologram_dim, , );		// number of pixels in each element hologram
rtDeclareVariable(uint2, num_rays_per_element_hologram, , );
rtDeclareVariable(uint2, half_num_rays_per_element_hologram, , );
rtDeclareVariable(float, pixel_pitch, , );
rtDeclareVariable(float, ray_interval, , );		// ray interval in radian
rtDeclareVariable(float3, hg_anchor, , );	// lower left corner of the hologram
rtDeclareVariable(float3, hg_v1, , );	// normalized
rtDeclareVariable(float3, hg_v2, , );	// normalized


rtDeclareVariable(unsigned int, radiance_ray_type, , );
rtDeclareVariable(float, scene_epsilon, , );
rtDeclareVariable(rtObject, top_object, , );


rtDeclareVariable(float3, eye, , );
rtDeclareVariable(float3, U, , );
rtDeclareVariable(float3, V, , );
rtDeclareVariable(float3, W, , );

/*
rtDeclareVariable(float4, m1, , );
rtDeclareVariable(float4, m2, , );
rtDeclareVariable(float4, m3, , );
rtDeclareVariable(float4, m4, , );
rtDeclareVariable(float4, n1, , );
rtDeclareVariable(float4, n2, , );
rtDeclareVariable(float4, n3, , );
rtDeclareVariable(float4, n4, , );
*/

rtDeclareVariable(float, fov, , );

RT_PROGRAM void camera() {
	// if outside buffer range, paint it black
	amplitude_buffer[launch_index] = make_float4(0.f, 0.f, 0.f, 0.0f);
	//location_buffer[launch_index] = make_float3(0.f, 0.f, 0.f);
	//initPhase_buffer[launch_index] = -1.f;
	size_t2 screen = amplitude_buffer.size();

	float2 d = (make_float2(launch_index) + make_float2(0.5f)) / make_float2(screen) * 2.f - 1.f;
	//rtPrintf("%f, %f\n", d.x, d.y);
	//d.x *= tanf(fov / 2.0f) * screen.x / (float)screen.y;
	//d.y *= 1* tanf(fov / 2.0f);
	//float3 angle = make_float3(cos(d.x) * sin(d.y), -cos(d.y), sin(d.x) * sin(d.y));
	//float3 ray_origin = make_float3(m4);
	float3 ray_origin = eye;
	float3 ray_direction = normalize(-d.x * (U) + -d.y * (V) + -(W));
	//float4 ray_dir = make_float4(d.x, -d.y, zNear, 1.0f);
	//ray_dir = ray_dir.x * n1 + ray_dir.y * n2 + ray_dir.z * n3 + ray_dir.w * n4;
	//ray_dir.z = -1.f;
	//ray_dir.w = 0.f;
	//rtPrintf("help: %f, %f, %f\n", ray_direction.x, ray_direction.y, ray_direction.z);
	//ray_dir = ray_dir.x * m1 + ray_dir.y * m2 + ray_dir.z * m3 + ray_dir.w * m4;
	//float3 ray_direction = normalize(make_float3(ray_dir/ray_dir.w) - ray_origin);
	//rtPrintf("view_project: %f, %f, %f\n", ray_direction);
	//rtPrintf("m1: %f, %f, %f\n", m1.x, m1.y, m1.z);
	//rtPrintf("m2: %f, %f, %f\n", m2.x, m2.y, m2.z);
	//rtPrintf("m3: %f, %f, %f\n\n", m3.x, m3.y, m3.z);
	//float3 ray_direction = normalize(make_float3((d.x * normalize(m1) + d.y * normalize(m2) + normalize(m3) + normalize(m4))));
	optix::Ray ray(ray_origin, ray_direction, radiance_ray_type, scene_epsilon);
	//rtPrintf("%f, %f, %f\n", rayDirection.x, rayDirection.y, rayDirection.z);
	PerRayData prd;
	rtTrace(top_object, ray, prd);
}







// exception program
// did not hit anything, mark as not hit
// zero amplitude,
RT_PROGRAM void exception() {
	rtPrintExceptionDetails();
	amplitude_buffer[launch_index] = make_float4(0.f, 1.f, 0.f, 1.0f);

}

// miss program
// did not hit anything, mark as not hit
// zero amplitude
RT_PROGRAM void miss() {
	//amplitude_buffer[launch_index] = optix::make_float4(0, 1, 1, 1);
}
