#include "hip/hip_runtime.h"
#include <optix.h>
#include <optixu/optixu_math_namespace.h>
#include <optix_device.h>

#define M_PIf 3.1415926535897932384626433

using namespace optix;

struct PerRayData_hologram {
	float3 f2b_color;
	float f2b_opaque;
	unsigned int depth;
};

rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );
rtDeclareVariable(float, t_hit, rtIntersectionDistance, );
rtDeclareVariable(uint2, launch_index, rtLaunchIndex, );

rtBuffer<float3, 2> location_buffer;
rtBuffer<float4, 2> amplitude_buffer;
rtBuffer<float, 2> initPhase_buffer;

rtBuffer<float3, 3> compLocation_buffer; // store location of each composition
rtBuffer<float3, 3> compAmplitude_buffer;// store real color contribution at this composition location
rtBuffer<unsigned int, 2> compDepth_buffer;

rtDeclareVariable(uint2, compositionBufferRowRange, , );
rtDeclareVariable(uint3, compositionBufferSize, , );
rtDeclareVariable(float, opacity_correction, , );

rtDeclareVariable(int, volumeTextureId, , );
rtDeclareVariable(int, transferFunction_texId, , );
rtDeclareVariable(int, initPhaseTextureId, , );
rtDeclareVariable(float3, texcoord, attribute texcoord, );
rtDeclareVariable(float3, front_hit_point, attribute front_hit_point, );
rtDeclareVariable(float3, back_hit_point, attribute back_hit_point, );

// information about the volume
rtDeclareVariable(float3, box_min, , );	// opposite corners of the volume
rtDeclareVariable(float3, box_max, , );
rtDeclareVariable(float3, v1, , );		// edges of the plane in which a slice is put, has been scaled by 1/dot(v1, v1)
rtDeclareVariable(float3, v2, , );
rtDeclareVariable(float3, v3, , );
rtDeclareVariable(float2, vol_hg_dist, , );
// step size
rtDeclareVariable(float, volumeRaytraceStepSize, , );

rtDeclareVariable(uint2, buffer_range_beg, , );
rtDeclareVariable(uint2, buffer_range_end, , );
RT_PROGRAM void dummy() {
	//rtPrintf("%d, %d\n", launch_index.x, launch_index.y);
	amplitude_buffer[launch_index] = make_float4(.7, 0, .9, .6);
}

rtDeclareVariable(float3, hg_normal, , );	// normalized

RT_PROGRAM void closest_hit() {
	//float t = -dot(ray.origin, hg_normal) / dot(ray.direction, hg_normal);
	//float3 interLoc = ray.origin + t * ray.direction;
	//location_buffer[launch_index] = interLoc;
	// init phase
	const float3 fhp = front_hit_point;
	//float phase_u = dot(fhp - box_min, v1);
	//float phase_v = dot(fhp - box_min, v2);
	//initPhase_buffer[launch_index] = optix::rtTex2D<float>(initPhaseTextureId, phase_u, phase_v);
	// composite from front to back
	const float3 bhp = back_hit_point;
	unsigned int num_steps = floorf(sqrtf(dot(bhp - fhp, bhp - fhp)) / volumeRaytraceStepSize);
	float3 color_composited = make_float3(0.f, 0.f, 0.f);
	float opaque_composited = 0.f;
	//float opacities[300];
	//int counter = 0;
	//rtPrintf("%d\n", num_steps);
	for (unsigned int s = 0; s < num_steps; ++s) {
		float3 texPoint = fhp + (s + 0.5) * volumeRaytraceStepSize * ray.direction;
		
		float vol_u = dot(texPoint - box_min, v1);
		float vol_v = dot(texPoint - box_min, v2);
		float vol_w = dot(texPoint - box_min, v3);
		
		float volume_scalar = optix::rtTex3D<float>(volumeTextureId, vol_u, vol_v, vol_w);

		float plus_u, minus_u, plus_v, minus_v, plus_w, minus_w;
		float region_increment = volumeRaytraceStepSize / 5.0f;

		plus_u = optix::rtTex3D<float>(volumeTextureId, vol_u + region_increment, vol_v, vol_w);
		minus_u = optix::rtTex3D<float>(volumeTextureId, vol_u - region_increment, vol_v, vol_w);

		plus_v = optix::rtTex3D<float>(volumeTextureId, vol_u, vol_v + region_increment, vol_w);
		minus_v = optix::rtTex3D<float>(volumeTextureId, vol_u, vol_v - region_increment, vol_w);

		plus_w = optix::rtTex3D<float>(volumeTextureId, vol_u, vol_v, vol_w + region_increment);
		minus_w = optix::rtTex3D<float>(volumeTextureId, vol_u, vol_v, vol_w - region_increment);

		float3 normal = optix::normalize(v1 * (plus_u - minus_u) / (2 * region_increment) + v2 * (plus_v - minus_v) / (2 * region_increment) + v3 * (plus_w - minus_w) / (2 * region_increment));

		float4 voxel_val_tf = optix::rtTex2D<float4>(transferFunction_texId, volume_scalar, volume_scalar);
		float3 color_self = normal;//make_float3(voxel_val_tf);
	
		
		float opaque_self = voxel_val_tf.w;
		if (opaque_self > .01) {
			opaque_self = 1.0f;
		}
		//opaque_self = 1.f - powf(1.f - opaque_self, opacity_correction);
		// amp = amp_in + (1-opaque) * amp_self
		// opqaue = opaque_in + (1-opqaue) * opqaue_self
		color_composited = color_composited + (1.f - opaque_composited) * opaque_self * color_self;
		opaque_composited = opaque_composited + (1.f - opaque_composited) * opaque_self;
		//if (counter	 < 300 && opaque_composited > 1e-3 ) {
			//opacities[counter] = opaque_self;
			//counter++;
		//}
		
		if (opaque_composited > 0.99) break;
	}
	if (opaque_composited > .9) {
		//rtPrintf("%f\n", opaque_composited);
		//for (int i = 0; i < counter; i++) {
			//rtPrintf("%f\n", opacities[i]);
		//}
		//rtPrintf("END\n");
	}
	amplitude_buffer[launch_index] = make_float4(sqrt(color_composited.x), sqrt(color_composited.y), sqrt(color_composited.z), (opaque_composited));
}


rtDeclareVariable(uint2, element_hologram_dim, , );		// number of pixels in each element hologram
rtDeclareVariable(uint2, num_rays_per_element_hologram, , );
rtDeclareVariable(uint2, half_num_rays_per_element_hologram, , );
rtDeclareVariable(float, pixel_pitch, , );
rtDeclareVariable(float, ray_interval, , );		// ray interval in radian
rtDeclareVariable(float3, hg_anchor, , );	// lower left corner of the hologram
rtDeclareVariable(float3, hg_v1, , );	// normalized
rtDeclareVariable(float3, hg_v2, , );	// normalized


rtDeclareVariable(unsigned int, radiance_ray_type, , );
rtDeclareVariable(float, scene_epsilon, , );
rtDeclareVariable(rtObject, top_object, , );

// ray direction jitter random texture
rtDeclareVariable(int, random_texture, , );	// [0, 1] ^ 4

rtDeclareVariable(float3, eye, , );
rtDeclareVariable(float3, U, , );
rtDeclareVariable(float3, V, , );
rtDeclareVariable(float3, W, , );

RT_PROGRAM void camera() {
	// if outside buffer range, paint it black
	amplitude_buffer[launch_index] = make_float4(0.f, 0.f, 0.f, 0.0f);
	//location_buffer[launch_index] = make_float3(0.f, 0.f, 0.f);
	//initPhase_buffer[launch_index] = -1.f;
	size_t2 screen = amplitude_buffer.size();

	float2 d = make_float2(launch_index) / make_float2(screen) * 2.f - 1.f;
	//float3 angle = make_float3(cos(d.x) * sin(d.y), -cos(d.y), sin(d.x) * sin(d.y));
	float3 ray_origin = eye;
	float3 ray_direction = normalize(d.x * -normalize(U) + d.y * -normalize(V) +  -normalize(W));

	optix::Ray ray(ray_origin, ray_direction, radiance_ray_type, scene_epsilon);
	//rtPrintf("%f, %f, %f\n", rayDirection.x, rayDirection.y, rayDirection.z);
	PerRayData_hologram prd;
	prd.f2b_color = make_float3(0.f, 0.f, 0.f);
	prd.f2b_opaque = 0.f;
	prd.depth = 0;
	rtTrace(top_object, ray, prd);
}

rtDeclareVariable(float, fov, , );




// Ray generation program
RT_PROGRAM void RayGeneration() {
	// if outside buffer range, paint it black
	amplitude_buffer[launch_index] = make_float4(0.f, 0.f, 0.f, 0.0f);
	location_buffer[launch_index] = make_float3(0.f, 0.f, 0.f);
	initPhase_buffer[launch_index] = -1.f;
	/*
	if (launch_index.x < buffer_range_beg.x || launch_index.x >= buffer_range_end.x ||
		launch_index.y < buffer_range_beg.y || launch_index.y >= buffer_range_end.y) {
		return;
	}
	if (compositionBufferRowRange.x < compositionBufferRowRange.y) {	// only trace a subregion
		uint launch_index_row_offset = launch_index.y - buffer_range_beg.y;	// offset in elemental hologram
		if (launch_index_row_offset < compositionBufferRowRange.x || launch_index_row_offset >= compositionBufferRowRange.y)
			return;
		uint2 compDepth_idx = make_uint2(launch_index.x - buffer_range_beg.x, launch_index_row_offset - compositionBufferRowRange.x);
		compDepth_buffer[compDepth_idx] = 0;
	}
	*/
	// 1. determine element hologram
	uint2 element_hologram_index = make_uint2(floor(make_float2(launch_index) / make_float2(num_rays_per_element_hologram)));
	// 2. determine center of element hologram
	float2 element_hologram_size = pixel_pitch * make_float2(element_hologram_dim);
	float2 ele_hg_center_from_anchor = (make_float2(element_hologram_index) + make_float2(0.5f, 0.5f)) * element_hologram_size;	// physical distance
	float3 ele_hg_center = hg_anchor + hg_v1 * ele_hg_center_from_anchor.x + hg_v2 * ele_hg_center_from_anchor.y;
	//rtPrintf("%f, %f, %f\n", ele_hg_center.x, ele_hg_center.y, ele_hg_center.z);
	// 3. determine vertical and horizontal angle
	float2 ray_angle = (make_float2(launch_index - element_hologram_index * num_rays_per_element_hologram) - make_float2(half_num_rays_per_element_hologram)) * ray_interval;
	// add random shift in ray direction
	float2 uv = make_float2(launch_index) / make_float2(amplitude_buffer.size());
	float2 ray_angle_shift = 2.f * make_float2(optix::rtTex2D<float4>(random_texture, uv.x, uv.y)) - 1.f;	// [-1, 1] x [-1, 1]
	//ray_angle = ray_angle + ray_angle_shift * ray_interval * 0.5f;
	// 4. determine ray direction
	float3 v_n1 = normalize(sinf(ray_angle.x) * hg_v1 + cosf(ray_angle.x) * hg_normal);
	//rtPrintf("ray_angle: %f, %f\nhg_v2: %f\nhg_n1: %f\n", ray_angle.x, ray_angle.y, hg_v2, v_n1);
	float3 ray_direction = normalize(sinf(ray_angle.y) * hg_v2 + cosf(ray_angle.y) * v_n1);
	//rtPrintf("Ray_Direction: %f, %f, %f\n", ray_direction.x, ray_direction.y, ray_direction.z);
	// ray

	optix::Ray ray(make_float3(0,0,-50), ray_direction, radiance_ray_type, scene_epsilon);
	PerRayData_hologram prd;
	prd.f2b_color = make_float3(0.f, 0.f, 0.f);
	prd.f2b_opaque = 0.f;
	prd.depth = 0;
	// trace
	//rtPrintf("%f, %f, %f\n", ray.origin.x,ray.origin.y, ray.origin.z );
	//rtPrintf("%f, %f, %f\n", ray.direction.x,ray.direction.y, ray.direction.z );
	rtTrace(top_object, ray, prd);
	// collect result
}

// exception program
// did not hit anything, mark as not hit
// zero amplitude,
RT_PROGRAM void exception() {
	rtPrintExceptionDetails();
	amplitude_buffer[launch_index] = make_float4(0.f, 1.f, 0.f, 1.0f);
	location_buffer[launch_index] = make_float3(0.f, 0.f, 0.f);
	initPhase_buffer[launch_index] = -1.f;
}

// miss program
// did not hit anything, mark as not hit
// zero amplitude
RT_PROGRAM void miss() {
	//amplitude_buffer[launch_index] = optix::make_float4(0, 1, 1, 1);
}
