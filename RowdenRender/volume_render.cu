#include "hip/hip_runtime.h"
#include <optix.h>
#include <optixu/optixu_math_namespace.h>
#include <optix_device.h>



using namespace optix;

struct PerRayData_hologram {
	float3 f2b_color;
	float f2b_opaque;
	unsigned int depth;
};

rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );
rtDeclareVariable(float, t_hit, rtIntersectionDistance, );
rtDeclareVariable(uint2, launch_index, rtLaunchIndex, );

rtBuffer<float4, 2> amplitude_buffer;


rtDeclareVariable(int, volumeTextureId1, , );
rtDeclareVariable(int, normalTextureId1, , );
rtDeclareVariable(int, volumeTextureId2, , );
rtDeclareVariable(int, normalTextureId2, , );
rtDeclareVariable(int, transferFunction_texId, , );
rtDeclareVariable(int, random_texture, , );
rtDeclareVariable(int, depth_mask_id, , );

rtDeclareVariable(float3, texcoord, attribute texcoord, );
rtDeclareVariable(float3, front_hit_point, attribute front_hit_point, );
rtDeclareVariable(float3, back_hit_point, attribute back_hit_point, );

// information about the volume
rtDeclareVariable(float3, box_min, , );	// opposite corners of the volume
rtDeclareVariable(float3, box_max, , );
rtDeclareVariable(float3, v1, , );		// edges of the plane in which a slice is put, has been scaled by 1/dot(v1, v1)
rtDeclareVariable(float3, v2, , );
rtDeclareVariable(float3, v3, , );

// step size
rtDeclareVariable(float, volumeRaytraceStepSize, , );

// lighting_stuff

rtDeclareVariable(float, ambientStrength, , );
rtDeclareVariable(float2, lightDirP, , );
rtDeclareVariable(float3, lightDir, , );
rtDeclareVariable(float2, sincosLightTheta, , );
rtDeclareVariable(float3, CameraDir, , );
rtDeclareVariable(float3, HalfwayVec, , );
rtDeclareVariable(float3, sincosCameraDirTheta, , );
rtDeclareVariable(float2, CamearDirP, , );
rtDeclareVariable(float2, HalfwayVecP, , );

rtDeclareVariable(float2, sincosHalfwayTheta, , );
rtDeclareVariable(float, specularStrength, , );
rtDeclareVariable(float, shininess, , );
rtDeclareVariable(float, diffuseStrength, , );

rtDeclareVariable(float, zFar, , );
rtDeclareVariable(float, zNear, , );

RT_PROGRAM void dummy() {
	//rtPrintf("%d, %d\n", launch_index.x, launch_index.y);
	amplitude_buffer[launch_index] = make_float4(.7, 0, .9, .6);
}



rtDeclareVariable(float3, hg_normal, , );	// normalized
//For use on two vectors with radius 1 in spherical coordinates
inline float sdot(float2 a, float2 b) {
	return  sin(a.x) * sin(b.x) * cos(a.y - b.y) + cos(a.x) * cos(b.x);
}
//For use on a sincos vector and the normal vector. Uses precomputed sines and cosines
inline float sdot(float2 sincosa, float2 sincosnorm, float a, float phi) {
	return sincosa.x * sincosnorm.x * cos(a - phi) + sincosa.y * sincosnorm.y;
}

RT_PROGRAM void closest_hit() {
	float2 sample;
	/*
	float max_theta = -3.15f;
	float min_theta = 3.15f;
	for (int i = 0; i < 512; i++) {
		for (int j = 0; j < 512; j++) {
			for (int k = 0; k < 512; k++) {
				sample = optix::rtTex3D<float2>(normalTextureId1, i/512.0f, j/512.0f, k/512.0f);
				//float theta = sample.x * M_PIf;
				float theta = sample.y * M_PIf * 2 - M_PIf;
				if (theta < min_theta) {
					min_theta = theta;
				}
				else if (theta > max_theta) {
					max_theta = theta;
				}
			}
		}
	}
	rtPrintf("%f, %f\n", min_theta, max_theta);
	*/
	const float3 fhp = front_hit_point;
	
	const float3 bhp = back_hit_point;
	unsigned int num_steps = floorf(sqrtf(dot(bhp - fhp, bhp - fhp)) / volumeRaytraceStepSize);
	float3 color_composited = make_float3(0.f, 0.f, 0.f);
	float opaque_composited = 0.f;
	//float epsilon = .1f * volumeRaytraceStepSize;
	//bool show_spec = true;
	float depth = optix::rtTex2D<float>(depth_mask_id, launch_index.x / (float)amplitude_buffer.size().x, (amplitude_buffer.size().y - launch_index.y) / (float)amplitude_buffer.size().y) * 2.0f - 1;
	for (unsigned int s = 0; s < num_steps; ++s) {
		float3 texPoint = fhp + (s)*volumeRaytraceStepSize * ray.direction;// +(epsilon) * (optix::rtTex3D<float>(random_texture, launch_index.x / amplitude_buffer.size().x, launch_index.y / amplitude_buffer.size().y, s / num_steps));

		float vol_u = dot(texPoint - box_min, v1);
		float vol_v = dot(texPoint - box_min, v2);
		float vol_w = dot(texPoint - box_min, v3);
		float3 show = texPoint - box_min;

		float volume_scalar;
		

		
		float4 color;
		bool flag = false;
		int i = 0;
			switch (i) {
			case 0:
				sample = optix::rtTex3D<float2>(normalTextureId1, vol_u, vol_v, vol_w);
				volume_scalar = optix::rtTex3D<float>(volumeTextureId1, vol_u, vol_v, vol_w);
				color = make_float4(253/255.0f, 117/255.0f, 0/255.0f, 1.0f);
				//color = make_float4(0, 0, 1, 1.0f);
				break;
			case 1:
				sample = optix::rtTex3D<float2>(normalTextureId2, vol_u, vol_v, vol_w);
				volume_scalar = optix::rtTex3D<float>(volumeTextureId2, vol_u, vol_v, vol_w);
				color = make_float4(51 / 255.0f, 160 / 255.0f, 0 / 255.0f, .99f);
				break;
			}
			float top_val = .565;
			//float top_val = .215;
			float bottom_val = .555;
			//float bottom_val = .2;
			float4 voxel_val_tf;// = optix::rtTex2D<float4>(transferFunction_texId, volume_scalar, volume_scalar);

			if (volume_scalar < top_val && volume_scalar > bottom_val) {
				voxel_val_tf = color;
				//voxel_val_tf = make_float4(fabs(normal.x), fabs(normal.y), fabs(normal.z), .99);
				//rtPrintf("%f, %f, %f\n", voxel_val_tf.x, voxel_val_tf.y, voxel_val_tf.z);
			}
			else {
				//voxel_val_tf = make_float4(0, 0, 0, 0);
				continue;
			}
			float phi = sample.x * M_PIf;
			float theta = sample.y * M_PIf * 2 - M_PIf;
			
			float distance = 0;
			//if ((zFar + zNear - depth * (zFar - zNear)) > 0) {
			distance = (2.0 * zNear * zFar) / (zFar + zNear - depth * (zFar - zNear));
			

			float3 color_self = make_float3(0);
			float opaque_self = 0;
			
			float sinphi = sin(phi);
			//float3 normal = make_float3(sinphi * cos(theta), sinphi * sin(theta), cos(phi));
			float2 normalP = make_float2(phi, theta);
			float2 sincosnorm = make_float2(sin(phi), cos(phi));
			
			//sin(theta1)sin(theta2)cos(phi1 - pih2) + cos(theta1)cos(theta2)
			//float diffuse = diffuseStrength * fmax(0, sin(theta) * sincosLightTheta.x * cos(phi - lightDirP.x) + cos(theta) * sincosLightTheta.y);
			//float diffuse = diffuseStrength * fmax(0, sdot(lightDirP, normalP));
			float diffuse = diffuseStrength * fmax(0, sdot(sincosLightTheta, sincosnorm, lightDirP.y, theta));
			//float3 viewDir = CameraDir;
				
				
				
				
			//float spec = specularStrength * pow(fmax(sin(theta) * sincosHalfwayTheta.x * cos(phi - sincosHalfwayTheta.x) + cos(theta) * sincosHalfwayTheta.y, 0), shininess);
			//float spec = specularStrength * pow(fabs(sdot(normalP, HalfwayVecP)), shininess);
			float spec = specularStrength * pow(fabs(sdot(sincosHalfwayTheta, sincosnorm, theta, HalfwayVecP.y)), shininess);
			//rtPrintf("%f, %f\n", HalfwayVecP.x, HalfwayVecP.y);
			color_self = ambientStrength * make_float3(voxel_val_tf) + diffuse * make_float3(voxel_val_tf) + spec * make_float3(1, 1, 1);
			//color_self = make_float3(fabs(normal.x), fabs(normal.y), fabs(normal.z));
				
			float bubble_coefficient = 1;// -(fabs(dot(cameraDir, normal));
				
			float top = .99;
			float bottom = .9;
			float max_oppac = .2f;
			float min_oppac = .02f;
				
			if (voxel_val_tf.w > 1e-5	) {
				if (bubble_coefficient > top) {
					bubble_coefficient = top;
				}
				else if (bubble_coefficient < bottom) {
					bubble_coefficient = bottom;
				}

				float norm = (((bubble_coefficient - bottom) / (top - bottom)));
				bubble_coefficient = (norm * (max_oppac - min_oppac)) + min_oppac;
				//voxel_val_tf.w = 0.05f;

				voxel_val_tf.w = 1.0f;
			}
			//bubble_coefficient /= 4;
			//bubble_coefficient += .75;
			//bubble_coefficient = 0 ;
			//if(spec != 0)
				//rtPrintf("%f\n", spec);
			float weight = .5;
			opaque_self = voxel_val_tf.w;// +((0 * bubble_coefficient) + ((weight)*spec));// 0.5f);
			

			/*
			if (opaque_self > .01) {
				opaque_self = 1.0f;
				//rtPrintf("%f, %f, %f\n", normal.x, normal.y, normal.z);
				if (fabs(normal.x) < 0.01 && fabs(normal.y) < 0.01 && fabs(normal.z) < 0.01) {
					opaque_self = 0;
				}
			}
			*/
			//opaque_self = 1.f - powf(1.f - opaque_self, opacity_correction);
			// amp = amp_in + (1-opaque) * amp_self
			// opqaue = opaque_in + (1-opqaue) * opqaue_self
			//rtPrintf("%f, %f, %f\n", color_self.x, color_self.y, color_self.z);
			color_composited = (color_composited + (1.f - opaque_composited) * color_self * opaque_self);
			opaque_composited = opaque_composited + (1.f - opaque_composited) * opaque_self;
			//color_composited = color_self;
			
			
			//if (counter	 < 300 && opaque_composited > 1e-3 ) {
				//opacities[counter] = opaque_self;
				//counter++;
			//}
			if (length(texPoint-ray.origin) > distance) {
				//color_composited = make_float3(1, 0, 0);
				//color_composited = make_float3(0, 0, 0);
				//opaque_composited = 0;
				//flag = true;
				//rtPrintf("%f\n", length(texPoint-box_min));
				//amplitude_buffer[launch_index] = make_float4(length(texPoint-ray.origin)/50.0f, length(texPoint-ray.origin) /50.0f, length(texPoint-ray.origin) /50.0f, 1.0f);
				//break;
			}
			if (opaque_composited > 0.99) break;
	}
	
	amplitude_buffer[launch_index] = make_float4((color_composited.x), (color_composited.y), (color_composited.z), (opaque_composited));
}


rtDeclareVariable(uint2, element_hologram_dim, , );		// number of pixels in each element hologram
rtDeclareVariable(uint2, num_rays_per_element_hologram, , );
rtDeclareVariable(uint2, half_num_rays_per_element_hologram, , );
rtDeclareVariable(float, pixel_pitch, , );
rtDeclareVariable(float, ray_interval, , );		// ray interval in radian
rtDeclareVariable(float3, hg_anchor, , );	// lower left corner of the hologram
rtDeclareVariable(float3, hg_v1, , );	// normalized
rtDeclareVariable(float3, hg_v2, , );	// normalized


rtDeclareVariable(unsigned int, radiance_ray_type, , );
rtDeclareVariable(float, scene_epsilon, , );
rtDeclareVariable(rtObject, top_object, , );


rtDeclareVariable(float3, eye, , );
rtDeclareVariable(float3, U, , );
rtDeclareVariable(float3, V, , );
rtDeclareVariable(float3, W, , );

rtDeclareVariable(float3, m1, , );
rtDeclareVariable(float3, m2, , );
rtDeclareVariable(float3, m3, , );
rtDeclareVariable(float4, m4, , );



RT_PROGRAM void camera() {
	// if outside buffer range, paint it black
	amplitude_buffer[launch_index] = make_float4(0.f, 0.f, 0.f, 0.0f);
	//location_buffer[launch_index] = make_float3(0.f, 0.f, 0.f);
	//initPhase_buffer[launch_index] = -1.f;
	size_t2 screen = amplitude_buffer.size();

	float2 d = make_float2(launch_index) / make_float2(screen) * 2.f - 1.f;
	//float3 angle = make_float3(cos(d.x) * sin(d.y), -cos(d.y), sin(d.x) * sin(d.y));
	float3 ray_origin = eye;
	float3 ray_direction = normalize(-d.x * (U) + -d.y * (V) + -(W));
	//rtPrintf("U: %f, %f, %f; %f, %f, %f\n", U.x, U.y, U.z, m1.x, m1.y, m1.z);
	//rtPrintf("V: %f, %f, %f; %f, %f, %f\n", V.x, V.y, V.z, m2.x, m2.y, m2.z);
	//rtPrintf("W: %f, %f, %f; %f, %f, %f\n", W.x, W.y, W.z, m3.x, m3.y, m3.z);
	//float3 ray_direction = normalize(make_float3((d.x * normalize(m1) + d.y * normalize(m2) + normalize(m3) + normalize(m4))));
	optix::Ray ray(ray_origin, ray_direction, radiance_ray_type, scene_epsilon);
	//rtPrintf("%f, %f, %f\n", rayDirection.x, rayDirection.y, rayDirection.z);
	PerRayData_hologram prd;
	prd.f2b_color = make_float3(0.f, 0.f, 0.f);
	prd.f2b_opaque = 0.f;
	prd.depth = 0;
	rtTrace(top_object, ray, prd);
}

rtDeclareVariable(float, fov, , );





// exception program
// did not hit anything, mark as not hit
// zero amplitude,
RT_PROGRAM void exception() {
	rtPrintExceptionDetails();
	amplitude_buffer[launch_index] = make_float4(0.f, 1.f, 0.f, 1.0f);

}

// miss program
// did not hit anything, mark as not hit
// zero amplitude
RT_PROGRAM void miss() {
	//amplitude_buffer[launch_index] = optix::make_float4(0, 1, 1, 1);
}
