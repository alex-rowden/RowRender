#include "hip/hip_runtime.h"
#include <optix.h>
#include <optixu/optixu_math_namespace.h>
#include <optix_device.h>

#define M_PIf 3.1415926535897932384626433

using namespace optix;

struct PerRayData_hologram {
	float3 f2b_color;
	float f2b_opaque;
	unsigned int depth;
};

rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );
rtDeclareVariable(float, t_hit, rtIntersectionDistance, );
rtDeclareVariable(uint2, launch_index, rtLaunchIndex, );

rtBuffer<float4, 2> amplitude_buffer;


rtDeclareVariable(int, volumeTextureId, , );
rtDeclareVariable(int, transferFunction_texId, , );
rtDeclareVariable(int, depth_mask_id, , );

rtDeclareVariable(float3, texcoord, attribute texcoord, );
rtDeclareVariable(float3, front_hit_point, attribute front_hit_point, );
rtDeclareVariable(float3, back_hit_point, attribute back_hit_point, );

// information about the volume
rtDeclareVariable(float3, box_min, , );	// opposite corners of the volume
rtDeclareVariable(float3, box_max, , );
rtDeclareVariable(float3, v1, , );		// edges of the plane in which a slice is put, has been scaled by 1/dot(v1, v1)
rtDeclareVariable(float3, v2, , );
rtDeclareVariable(float3, v3, , );

// step size
rtDeclareVariable(float, volumeRaytraceStepSize, , );

// lighting_stuff

rtDeclareVariable(float, ambientStrength, , );
rtDeclareVariable(float3, lightPos, , );
rtDeclareVariable(float, specularStrength, , );
rtDeclareVariable(float, shininess, , );

rtDeclareVariable(float, zFar, , );
rtDeclareVariable(float, zNear, , );

RT_PROGRAM void dummy() {
	//rtPrintf("%d, %d\n", launch_index.x, launch_index.y);
	amplitude_buffer[launch_index] = make_float4(.7, 0, .9, .6);
}

rtDeclareVariable(float3, hg_normal, , );	// normalized

RT_PROGRAM void closest_hit() {
	//float t = -dot(ray.origin, hg_normal) / dot(ray.direction, hg_normal);
	//float3 interLoc = ray.origin + t * ray.direction;
	//location_buffer[launch_index] = interLoc;
	// init phase
	const float3 fhp = front_hit_point;
	//float phase_u = dot(fhp - box_min, v1);
	//float phase_v = dot(fhp - box_min, v2);
	//initPhase_buffer[launch_index] = optix::rtTex2D<float>(initPhaseTextureId, phase_u, phase_v);
	// composite from front to back
	const float3 bhp = back_hit_point;
	unsigned int num_steps = floorf(sqrtf(dot(bhp - fhp, bhp - fhp)) / volumeRaytraceStepSize);
	float3 color_composited = make_float3(0.f, 0.f, 0.f);
	float opaque_composited = 0.f;
	//float opacities[300];
	//int counter = 0;
	//rtPrintf("%d\n", num_steps);
	for (unsigned int s = 0; s < num_steps; ++s) {
		float3 texPoint = fhp + (s + 0.5) * volumeRaytraceStepSize * ray.direction;
		
		float vol_u = dot(texPoint - box_min, v1);
		float vol_v = dot(texPoint - box_min, v2);
		float vol_w = dot(texPoint - box_min, v3);
		float3 show = texPoint - box_min;
		
		

		float volume_scalar = optix::rtTex3D<float>(volumeTextureId, vol_u, vol_v, vol_w);

		float plus_u, minus_u, plus_v, minus_v, plus_w, minus_w;
		float region_increment = volumeRaytraceStepSize / 5.0f;

		plus_u = optix::rtTex3D<float>(volumeTextureId, vol_u + region_increment, vol_v, vol_w);
		minus_u = optix::rtTex3D<float>(volumeTextureId, vol_u - region_increment, vol_v, vol_w);

		plus_v = optix::rtTex3D<float>(volumeTextureId, vol_u, vol_v + region_increment, vol_w);
		minus_v = optix::rtTex3D<float>(volumeTextureId, vol_u, vol_v - region_increment, vol_w);

		plus_w = optix::rtTex3D<float>(volumeTextureId, vol_u, vol_v, vol_w + region_increment);
		minus_w = optix::rtTex3D<float>(volumeTextureId, vol_u, vol_v, vol_w - region_increment);

		float3 normal = -optix::normalize(normalize(v1) * (plus_u - minus_u) / (2 * region_increment) + normalize(v2) * (plus_v - minus_v) / (2 * region_increment) + normalize(v3) * (plus_w - minus_w) / (2 * region_increment));

		float4 voxel_val_tf = optix::rtTex2D<float4>(transferFunction_texId, volume_scalar, volume_scalar);
		//float3 color_self = make_float3(fabs(normal.x), fabs(normal.y), fabs(normal.z));//make_float3(voxel_val_tf);
		float3 lightDir = (lightPos - (texPoint - box_min));
		
		//float depth = optix::rtTex2D<float>(depth_mask_id, vol_u, vol_v) * 2.0f - 1;
		//if (depth != -1 && depth != 1) {
			//rtPrintf("%f\n", depth);
		//}
		//if(depth < 1)
			//rtPrintf("%f\n", depth);
		//float distance = 10000;
		//if ((zFar + zNear - depth * (zFar - zNear)) > 0) {
		//	distance = (2.0 * zNear * zFar) / (zFar + zNear - depth * (zFar - zNear));
		//	//rtPrintf("%f, %f\n", texPoint.z, distance);
		//}
		//rtPrintf("%f\n", (texPoint-box_min).z);
		//if ((texPoint-box_min).z < distance * 25.0) {
		//	break;
		//}
		
		float3 color_self = make_float3(0);
		float opaque_self = 0;
		//rtPrintf("%f\n", lightDir.x * lightDir.x + lightDir.y * lightDir.y + lightDir.z * lightDir.z);
		if (lightDir.x * lightDir.x + lightDir.y * lightDir.y + lightDir.z * lightDir.z < -1) {
			color_self = make_float3(0, 1, 0);
			opaque_self = 1;
		}
		else {
			lightDir = normalize(lightDir);
			float diffuse = fmax(dot(normal, lightDir), 0.0f);

			float3 viewDir = normalize(ray.origin - (texPoint));
			float3 reflectDir = reflect(-lightDir, normal);
			float spec = specularStrength * pow(fmax(dot(viewDir, reflectDir), 0.0f), shininess);
			color_self = (ambientStrength + diffuse) * make_float3(voxel_val_tf) + spec * make_float3(1, 1, 1);
			opaque_self = voxel_val_tf.w;
		}
		
		/*
		if (opaque_self > .01) {
			opaque_self = 1.0f;
			//rtPrintf("%f, %f, %f\n", normal.x, normal.y, normal.z);
			if (fabs(normal.x) < 0.01 && fabs(normal.y) < 0.01 && fabs(normal.z) < 0.01) {
				opaque_self = 0;
			}
		}
		*/
		//opaque_self = 1.f - powf(1.f - opaque_self, opacity_correction);
		// amp = amp_in + (1-opaque) * amp_self
		// opqaue = opaque_in + (1-opqaue) * opqaue_self
		color_composited = color_composited + (1.f - opaque_composited) * opaque_self * color_self;
		opaque_composited = opaque_composited + (1.f - opaque_composited) * opaque_self;
		//if (counter	 < 300 && opaque_composited > 1e-3 ) {
			//opacities[counter] = opaque_self;
			//counter++;
		//}
		
		if (opaque_composited > 0.99) break;
	}
	if (opaque_composited > .9) {
		//rtPrintf("%f\n", opaque_composited);
		//for (int i = 0; i < counter; i++) {
			//rtPrintf("%f\n", opacities[i]);
		//}
		//rtPrintf("END\n");
	}
	amplitude_buffer[launch_index] = make_float4(sqrt(color_composited.x), sqrt(color_composited.y), sqrt(color_composited.z), (opaque_composited));
}


rtDeclareVariable(uint2, element_hologram_dim, , );		// number of pixels in each element hologram
rtDeclareVariable(uint2, num_rays_per_element_hologram, , );
rtDeclareVariable(uint2, half_num_rays_per_element_hologram, , );
rtDeclareVariable(float, pixel_pitch, , );
rtDeclareVariable(float, ray_interval, , );		// ray interval in radian
rtDeclareVariable(float3, hg_anchor, , );	// lower left corner of the hologram
rtDeclareVariable(float3, hg_v1, , );	// normalized
rtDeclareVariable(float3, hg_v2, , );	// normalized


rtDeclareVariable(unsigned int, radiance_ray_type, , );
rtDeclareVariable(float, scene_epsilon, , );
rtDeclareVariable(rtObject, top_object, , );

// ray direction jitter random texture
rtDeclareVariable(int, random_texture, , );	// [0, 1] ^ 4

rtDeclareVariable(float3, eye, , );
rtDeclareVariable(float3, U, , );
rtDeclareVariable(float3, V, , );
rtDeclareVariable(float3, W, , );

RT_PROGRAM void camera() {
	// if outside buffer range, paint it black
	amplitude_buffer[launch_index] = make_float4(0.f, 0.f, 0.f, 0.0f);
	//location_buffer[launch_index] = make_float3(0.f, 0.f, 0.f);
	//initPhase_buffer[launch_index] = -1.f;
	size_t2 screen = amplitude_buffer.size();

	float2 d = make_float2(launch_index) / make_float2(screen) * 2.f - 1.f;
	//float3 angle = make_float3(cos(d.x) * sin(d.y), -cos(d.y), sin(d.x) * sin(d.y));
	float3 ray_origin = eye;
	float3 ray_direction = normalize(d.x * -normalize(U) + d.y * -normalize(V) +  -normalize(W));

	optix::Ray ray(ray_origin, ray_direction, radiance_ray_type, scene_epsilon);
	//rtPrintf("%f, %f, %f\n", rayDirection.x, rayDirection.y, rayDirection.z);
	PerRayData_hologram prd;
	prd.f2b_color = make_float3(0.f, 0.f, 0.f);
	prd.f2b_opaque = 0.f;
	prd.depth = 0;
	rtTrace(top_object, ray, prd);
}

rtDeclareVariable(float, fov, , );





// exception program
// did not hit anything, mark as not hit
// zero amplitude,
RT_PROGRAM void exception() {
	rtPrintExceptionDetails();
	amplitude_buffer[launch_index] = make_float4(0.f, 1.f, 0.f, 1.0f);

}

// miss program
// did not hit anything, mark as not hit
// zero amplitude
RT_PROGRAM void miss() {
	//amplitude_buffer[launch_index] = optix::make_float4(0, 1, 1, 1);
}
