#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2018, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include <optix.h>
#include <optixu/optixu_math_namespace.h>
#include <optixu/optixu_matrix_namespace.h>
#include <optixu/optixu_aabb_namespace.h>
#include <optix_device.h>
using namespace optix;


rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );


rtDeclareVariable(float3, box_min, , );	// also the anchor
rtDeclareVariable(float3, box_max, , );	// opposite corners of the volume
rtDeclareVariable(float3, v1, , );		// edges of the plane in which a slice is put, has been scaled by 1/dot(v1, v1)
rtDeclareVariable(float3, v2, , );
rtDeclareVariable(float3, v3, , );
rtDeclareVariable(float3, voxel_size, , );
rtDeclareVariable(float, scene_epsilon, , );
rtDeclareVariable(float, volumeRaytraceStepSize, , );

rtDeclareVariable(float3, texcoord, attribute texcoord, );
rtDeclareVariable(float3, front_hit_point, attribute front_hit_point, );
rtDeclareVariable(float3, back_hit_point, attribute back_hit_point, );



RT_PROGRAM void box_intersect(int primIdx) {
	float3 rayOrigin_boxMin = box_min - ray.origin;
	float3 rayOrigin_boxMax = box_max - ray.origin;
	float3 t0 = rayOrigin_boxMin / ray.direction;
	float3 t1 = rayOrigin_boxMax / ray.direction;
	float3 near = fminf(t0, t1);
	float3 far = fmaxf(t0, t1);
	float t_min = fmaxf(near);
	float t_max = fminf(far);

	if (t_min < t_max) {

		bool check_second = true;
		// ray intersects volume, enters at t_min, exits at t_max
		if (rtPotentialIntersection(t_min)) {
			check_second = false;
			//rtPrintf("%f, %f\n", t_min, t_max);
			front_hit_point = ray.origin + (t_min + scene_epsilon) * ray.direction;
			back_hit_point = ray.origin + (t_max - scene_epsilon) * ray.direction;
			rtReportIntersection(0);
		}
		if (check_second) {
			if (rtPotentialIntersection(t_max)) {
				front_hit_point = ray.origin + ( scene_epsilon)* ray.direction;
				back_hit_point = ray.origin + (t_max - scene_epsilon) * ray.direction;
				rtReportIntersection(0);
			}
		}

	}
}
RT_PROGRAM void box_bounds(int primIdx, float result[6])
{
	optix::Aabb* aabb = (optix::Aabb*)result;
	//rtPrintf("%f, %f, %f\n", box_min.x, box_min.y, box_min.z);
	//rtPrintf("%f, %f, %f\n", box_max.x, box_max.y, box_max.z);
	aabb->set(box_min, box_max);
}
