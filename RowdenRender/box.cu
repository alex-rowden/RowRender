#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2018, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include <optix.h>
#include <optixu/optixu_math_namespace.h>
#include <optixu/optixu_matrix_namespace.h>
#include <optixu/optixu_aabb_namespace.h>

using namespace optix;

rtBuffer<float4> voxel_buffer;
rtBuffer<float> intensity_buffer;

rtDeclareVariable(float, cutoff_to, , );
rtDeclareVariable(float, cutoff_from, , );
rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );
rtDeclareVariable(float3, geometric_normal, attribute geometric_normal, );
rtDeclareVariable(float3, shading_normal, attribute shading_normal, );
rtDeclareVariable(float4, obj_color, attribute obj_color, );


//
// Box
//
static __device__ void make_box(const float4& input, float3& boxmin, float3& boxmax) {
	float halfWidth = input.w / 2;
	boxmin.x = input.x - halfWidth; boxmax.x = input.x + halfWidth;
	boxmin.y = input.y - halfWidth; boxmax.y = input.y + halfWidth;
	boxmin.z = input.z - halfWidth; boxmax.z = input.z + halfWidth;
}

static __device__ float3 boxnormal(float t, float3 t0, float3 t1)
{
	float3 neg = make_float3(t == t0.x ? 1 : 0, t == t0.y ? 1 : 0, t == t0.z ? 1 : 0);
	float3 pos = make_float3(t == t1.x ? 1 : 0, t == t1.y ? 1 : 0, t == t1.z ? 1 : 0);
	return pos - neg;
}

static __device__ float4 get_color(float value) {
	if (value > .8) {
		return make_float4(1, 0, 1, .02);
	}
	else {
		return make_float4(0, 0, 0, 0);
	}
}

RT_PROGRAM void box_intersect(int idx)
{
	if (intensity_buffer[idx] == 0) return;
	else if (intensity_buffer[idx] < cutoff_from || intensity_buffer[idx] > cutoff_to) return;
	if (intensity_buffer[idx] < .8) return;

	float3 boxmin, boxmax;
	make_box(voxel_buffer[idx], boxmin, boxmax);

	float3 t0 = (boxmin - ray.origin) / ray.direction;
	float3 t1 = (boxmax - ray.origin) / ray.direction;

	float3 near = fminf(t0, t1);
	float3 far = fmaxf(t0, t1);
	float tmin = fmaxf(near);
	float tmax = fminf(far);

	if (tmin <= tmax) {
		bool check_second = true;
		if (rtPotentialIntersection(tmin)) {
			shading_normal = geometric_normal = boxnormal(tmin, t0, t1);
			obj_color = get_color(intensity_buffer[idx]);
			if (rtReportIntersection(0))
				check_second = false;
		}
		if (check_second) {
			if (rtPotentialIntersection(tmax)) {
				obj_color = get_color(intensity_buffer[idx]);
				shading_normal = geometric_normal = boxnormal(tmax, t0, t1);
				rtReportIntersection(0);
			}
		}
	}
}

RT_PROGRAM void box_bounds(int primIdx, float result[6])
{
	float3 boxmin, boxmax;
	make_box(voxel_buffer[primIdx], boxmin, boxmax);
	optix::Aabb* aabb = (optix::Aabb*)result;
	aabb->set(boxmin, boxmax);
}
